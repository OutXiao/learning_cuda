#include "hip/hip_runtime.h"
#include "arithmetic.cuh"

__global__ void gpu_arithmetic(real* d_x, const real x0, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        real x_tmp = d_x[n];
        while (sqrt(x_tmp) < x0)
        {
            ++x_tmp;
        }
        d_x[n] = x_tmp;
    }
}

void cpu_arithmetic(real* x, const real x0, const int N)
{
    for (int n = 0; n < N; ++n)
    {
        real x_tmp = x[n];
        while (sqrt(x_tmp) < x0)
        {
            ++x_tmp;
        }
        x[n] = x_tmp;
    }
}

hipError_t test_gpu_arithmetic()
{
    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    const int M = sizeof(real) * N;
    real* h_x = (real*)malloc(M);
    real* d_x;
    CHECK(hipMalloc((void**)&d_x, M));

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        for (int n = 0; n < N; ++n)
        {
            h_x[n] = 0.0;
        }
        CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));e
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        gpu_arithmetic << <grid_size, block_size >> > (d_x, x0, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    free(h_x);
    CHECK(hipFree(d_x));
    return hipSuccess;
}

hipError_t test_cpu_arithmetic()
{
    const int M = sizeof(real) * N;
    real* x = (real*)malloc(M);

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        for (int n = 0; n < N; ++n)
        {
            x[n] = 0.0;
        }

        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        cpu_arithmetic(x, x0, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    free(x);
    return hipSuccess;
}